#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <arpa/inet.h>
#include <netinet/ip.h>
#include <netinet/ip6.h>
#include <stddef.h>
#include <ixmap.h>
#include <ixmap_cuda.h>

extern "C" {
#include "linux/list_cuda.h"
#include "main.h"
#include "forward.h"
#include "thread.h"
#include "neigh.h"
#include "fib.h"
#include "hip/hip_runtime.h"
}

__global__ static void forward_process(struct ixmapfwd_thread *thread,
	unsigned int port_index, struct ixmap_packet *packet,
	struct ixmap_packet_cuda *result);
__device__ static int forward_ip_process(struct ixmapfwd_thread *thread,
	unsigned int port_index, struct ixmap_packet *packet);
__device__ static int forward_ip6_process(struct ixmapfwd_thread *thread,
	unsigned int port_index, struct ixmap_packet *packet);

__host__ void forward_process_offload(struct ixmapfwd_thread *thread,
	unsigned int port_index, struct ixmap_packet *packet,
	unsigned int num_packets)
{
	struct ixmap_packet_cuda result[IXMAP_RX_BUDGET];
	int fd, i;

	forward_process<<<CUDA_NMPROCS, CUDA_NTHREADS>>>
		(thread, port_index, packet, result);

	for(i = 0; i < num_packets; i++){
		if(result[i].outif >= 0){
			ixmap_tx_assign(thread->plane, result[i].outif,
				thread->buf, &packet[i]);
		}else if(result[i].outif == -1){
			goto packet_drop;
		}else{
			goto packet_inject;
		}

		continue;
packet_inject:
		fd = thread->tun_plane->ports[port_index].fd;
		write(fd, packet[i].slot_buf, packet[i].slot_size);
packet_drop:
		ixmap_slot_release(thread->buf, packet[i].slot_index);
	}
	return;
}

__global__ static void forward_process(struct ixmapfwd_thread *thread,
	unsigned int port_index, struct ixmap_packet *packet,
	struct ixmap_packet_cuda *result)
{
	struct ethhdr *eth;
	int index;

	index = blockIdx.x * threadIdx.x;

	eth = (struct ethhdr *)packet[index].slot_buf;
	switch(bswap_16(eth->h_proto)){
	case ETH_P_ARP:
		result[index].outif = -2;
		break;
	case ETH_P_IP:
		result[index].outif =
			forward_ip_process(thread, port_index, &packet[index]);
		break;
	case ETH_P_IPV6:
		result[index].outif =
			forward_ip6_process(thread, port_index, &packet[index]);
		break;
	default:
		result[index].outif = -1;
		break;
	}

	return;
}

__device__ static int forward_ip_process(struct ixmapfwd_thread *thread,
	unsigned int port_index, struct ixmap_packet *packet)
{
	struct ethhdr		*eth;
	struct iphdr		*ip;
	struct fib_entry	*fib_entry;
	struct neigh_entry	*neigh_entry;
	uint8_t			*dst_mac, *src_mac;
	uint32_t		check;
	int			ret;

	eth = (struct ethhdr *)packet->slot_buf;
	ip = (struct iphdr *)(packet->slot_buf + sizeof(struct ethhdr));

	fib_entry = fib_lookup(thread->fib_inet, &ip->daddr);
	if(!fib_entry)
		goto packet_drop;

	if(fib_entry->port_index < 0)
		goto packet_local;

	switch(fib_entry->type){
	case FIB_TYPE_LOCAL:
		goto packet_local;
		break;
	case FIB_TYPE_LINK:
		neigh_entry = neigh_lookup(
			thread->neigh_inet[fib_entry->port_index],
			&ip->daddr);
		break;
	case FIB_TYPE_FORWARD:
		neigh_entry = neigh_lookup(
			thread->neigh_inet[fib_entry->port_index],
			fib_entry->nexthop);
		break;
	default:
		neigh_entry = NULL;
		break;
	}

	if(!neigh_entry)
		goto packet_local;

	if(ip->ttl == 1)
		goto packet_local;

	ip->ttl--;

	check = ip->check;
	check += bswap_16(0x0100);
	ip->check = check + ((check >= 0xFFFF) ? 1 : 0);

	dst_mac = neigh_entry->dst_mac;
	src_mac = ixmap_macaddr_cuda(thread->plane, fib_entry->port_index);
	memcpy(eth->h_dest, dst_mac, ETH_ALEN);
	memcpy(eth->h_source, src_mac, ETH_ALEN);

	ret = fib_entry->port_index;
	return ret;

packet_local:
	return -2;
packet_drop:
	return -1;
}

__device__ static int forward_ip6_process(struct ixmapfwd_thread *thread,
	unsigned int port_index, struct ixmap_packet *packet)
{
	struct ethhdr		*eth;
	struct ip6_hdr		*ip6;
	struct fib_entry	*fib_entry;
	struct neigh_entry	*neigh_entry;
	uint8_t			*dst_mac, *src_mac;
	int			ret;

	eth = (struct ethhdr *)packet->slot_buf;
	ip6 = (struct ip6_hdr *)(packet->slot_buf + sizeof(struct ethhdr));

	if(ip6->ip6_dst.s6_addr[0] == 0xfe
	&& (ip6->ip6_dst.s6_addr[1] & 0xc0) == 0x80)
		goto packet_local;

	fib_entry = fib_lookup(thread->fib_inet6, (uint32_t *)&ip6->ip6_dst);
	if(!fib_entry)
		goto packet_drop;

	if(fib_entry->port_index < 0)
		goto packet_local;

	switch(fib_entry->type){
	case FIB_TYPE_LOCAL:
		goto packet_local;
		break;
	case FIB_TYPE_LINK:
		neigh_entry = neigh_lookup(
			thread->neigh_inet6[fib_entry->port_index],
			&ip6->ip6_dst);
		break;
	case FIB_TYPE_FORWARD:
		neigh_entry = neigh_lookup(
			thread->neigh_inet6[fib_entry->port_index],
			fib_entry->nexthop);
		break;
	default:
		neigh_entry = NULL;
		break;
	}

	if(!neigh_entry)
		goto packet_local;

	if(ip6->ip6_hlim == 1)
		goto packet_local;

	ip6->ip6_hlim--;

	dst_mac = neigh_entry->dst_mac;
	src_mac = ixmap_macaddr_cuda(thread->plane, fib_entry->port_index);
	memcpy(eth->h_dest, dst_mac, ETH_ALEN);
	memcpy(eth->h_source, src_mac, ETH_ALEN);

	ret = fib_entry->port_index;
	return ret;

packet_local:
	return -2;
packet_drop:
	return -1;
}

